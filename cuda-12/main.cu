#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <float.h>
#include <hiprand.h>
#include "vec3.h"
#include "ray.h"
#include "hitable.h"
#include "hitablelist.h"
#include "sphere.h"
#include "camera.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}
__global__ void create_world(hitable** d_list, hitable** d_world)
{
    if(threadIdx.x==0 && blockIdx.x==0)
    {
        *(d_list) = new sphere(vec3(0,0,-1), 0.5f);
        *(d_list+1) = new sphere(vec3(0,-100.5f,-1), 100);
        *d_world = new hitable_list(d_list, 2);
    }
}

__global__ void free_world(hitable **d_list, hitable **d_world)
{
    delete *(d_list);
    delete *(d_list+1);
    delete *d_world;
}

__global__ void create_camera(camera** cam)
{
    if(threadIdx.x==0 && blockIdx.x==0)
    {
        vec3 lookfrom(0.0, 0.0, 0.0);
        vec3 lookat(0.0, 0.0, -1.0);
        vec3 vup(0.0, 1.0, 0.0);
        float vfov = 90.0;
        float aspect = 2.0;
        float aperture = 0.0;
        float focus_dist = (lookat-lookfrom).length();
        *cam = new camera(lookfrom, lookat, vup, vfov, aspect, apreture, focus_dist);
    }
}

__global__ free_camera(camera** cam)
{
    if(threadIdx.x==0 && blockIdx.x==0)
        delete *(cam);
}
__device__ vec3 color(const ray& r, hitable** world)
{
    hit_record rec;
    if((*world)->hit(r, 0.0f, FLT_MAX, rec))
    {
        return 0.5f*vec3(rec.normal.x()+1.0f, rec.normal.y()+1.0f, rec.normal.z()+1.0f);
    }
    vec3 unit_direction = unit_vector(r.direction());
    float t = 0.5f*(unit_direction.y() + 1.0f);
    return (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state)
{

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3 *fb, int max_x, int max_y, int ns, camera ** cam, hitable** world, hiprandState *rand_state) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0,0,0);
    for(int s=0; s<ns; s++)
    {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u,v);
        col += color(r, world);
    }
    fb[pixel_index] = col/float(ns);
}


int main() {
    int nx = 1200;
    int ny = 600;
    int tx = 8;
    int ty = 8;

    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);

    vec3 lower_left_corner(-2.0, -1.0, -1.0);
    vec3 horizontal(4.0, 0.0, 0.0);
    vec3 vertical(0.0, 2.0, 0.0);
    vec3 origin(0.0, 0.0, 0.0);

    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx*ny;
    size_t fb_size = num_pixels*sizeof(vec3);

    // hiprand init
    hiprandState * d_rand_state;
    checkCudaErrors(hipMalloc((void**)&d_rand_state, num_pixels*sizeof(hiprandState)));

    // allocate FB
    vec3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // create world
    hitable **d_list;
    checkCudaErrors(hipMalloc((void**) &d_list, 2*sizeof(hitable*)));
    hitable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));
    create_world<<<1,1>>>(d_list,d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    //create camera
    camera **d_cam;
    checkCudaErrors(hipMalloc((void**) &d_cam, sizeof(camera*)));
    create_camera<<<1,1>>>(d_cam);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    // Render our buffer
    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads>>>(fb, nx, ny, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output FB as Image
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*nx + i;
            int ir = int(255.99*fb[pixel_index].r());
            int ig = int(255.99*fb[pixel_index].g());
            int ib = int(255.99*fb[pixel_index].b());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list, d_world);
    checkCudaErrors(hipGetLastError());
    free_camera<<<1,1>>>(cam);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(cam));
    checkCudaErrors(hipFree(fb));
}
